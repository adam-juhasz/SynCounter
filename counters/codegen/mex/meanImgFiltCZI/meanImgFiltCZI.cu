#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * meanImgFiltCZI.cu
 *
 * Code generation for function 'meanImgFiltCZI'
 *
 */

/* Include files */
#include "MWCudaDimUtility.h"
#include "rt_nonfinite.h"
#include "meanImgFiltCZI.h"

/* Function Declarations */
static __global__ void meanImgFiltCZI_kernel1(real_T expanded[1052676]);
static __global__ void meanImgFiltCZI_kernel2(const real_T A[1048576], real_T
  expanded[1052676]);
static __global__ void meanImgFiltCZI_kernel3(real_T expanded[1052676], real_T
  B[1048576]);

/* Function Definitions */
static __global__ __launch_bounds__(512, 1) void meanImgFiltCZI_kernel1(real_T
  expanded[1052676])
{
  uint32_T threadId;
  int32_T ocol;
  threadId = (uint32_T)mwGetGlobalThreadIndex();
  ocol = (int32_T)threadId;
  if (ocol < 1052676) {
    expanded[ocol] = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void meanImgFiltCZI_kernel2(const
  real_T A[1048576], real_T expanded[1052676])
{
  uint32_T threadId;
  int32_T k;
  int32_T ocol;
  threadId = (uint32_T)mwGetGlobalThreadIndex();
  k = (int32_T)(threadId % 1024U);
  ocol = (int32_T)((threadId - (uint32_T)k) / 1024U);
  if (ocol < 1024) {
    expanded[(k + 1026 * (1 + ocol)) + 1] = A[k + (ocol << 10)];
  }
}

static __global__ __launch_bounds__(1024, 1) void meanImgFiltCZI_kernel3(real_T
  expanded[1052676], real_T B[1048576])
{
  real_T y;
  int32_T orow;
  int32_T ocol;
  int32_T k;
  __shared__ real_T expanded_shared[1156];
  int32_T baseR;
  int32_T srow;
  int32_T strideRow;
  int32_T scol;
  int32_T strideCol;
  int32_T y_idx;
  int32_T baseC;
  int32_T x_idx;
  ocol = mwGetGlobalThreadIndexInYDimension();
  orow = mwGetGlobalThreadIndexInXDimension();
  baseR = orow;
  srow = (int32_T)threadIdx.x;
  strideRow = (int32_T)blockDim.x;
  scol = (int32_T)threadIdx.y;
  strideCol = (int32_T)blockDim.y;
  for (y_idx = srow; y_idx <= 33; y_idx += strideRow) {
    baseC = ocol;
    for (x_idx = scol; x_idx <= 33; x_idx += strideCol) {
      if (((int32_T)(((int32_T)(baseR >= 0)) && ((int32_T)(baseR < 1026)))) &&
          ((int32_T)(((int32_T)(baseC >= 0)) && ((int32_T)(baseC < 1026))))) {
        expanded_shared[y_idx + 34 * x_idx] = expanded[1026 * baseC + baseR];
      } else {
        expanded_shared[y_idx + 34 * x_idx] = 0.0;
      }

      baseC += strideCol;
    }

    baseR += strideRow;
  }

  __syncthreads();
  if (((int32_T)(ocol < 1024)) && ((int32_T)(orow < 1024))) {
    y = expanded_shared[((int32_T)threadIdx.x +
                         (mwGetGlobalThreadIndexInXDimension() -
                          mwGetGlobalThreadIndexInXDimension())) + 34 *
      ((int32_T)threadIdx.y + (mwGetGlobalThreadIndexInYDimension() -
        mwGetGlobalThreadIndexInYDimension()))];
    for (k = 0; k < 8; k++) {
      y += expanded_shared[((int32_T)threadIdx.x + (((k + 1) % 3 +
        mwGetGlobalThreadIndexInXDimension()) -
        mwGetGlobalThreadIndexInXDimension())) + 34 * ((int32_T)threadIdx.y +
        (((k + 1) / 3 + mwGetGlobalThreadIndexInYDimension()) -
         mwGetGlobalThreadIndexInYDimension()))];
    }

    B[orow + (ocol << 10)] = y / 9.0;
  }
}

void meanImgFiltCZI(const real_T A[1048576], real_T B[1048576])
{
  real_T (*gpu_expanded)[1052676];
  real_T (*gpu_A)[1048576];
  real_T (*gpu_B)[1048576];
  hipMalloc(&gpu_B, 8388608ULL);
  hipMalloc(&gpu_A, 8388608ULL);
  hipMalloc(&gpu_expanded, 8421408ULL);
  meanImgFiltCZI_kernel1<<<dim3(2057U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_expanded);
  hipMemcpy(gpu_A, (void *)&A[0], 8388608ULL, hipMemcpyHostToDevice);
  meanImgFiltCZI_kernel2<<<dim3(2048U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_A,
    *gpu_expanded);
  meanImgFiltCZI_kernel3<<<dim3(32U, 32U, 1U), dim3(32U, 32U, 1U)>>>
    (*gpu_expanded, *gpu_B);
  hipMemcpy(&B[0], gpu_B, 8388608ULL, hipMemcpyDeviceToHost);
  hipFree(*gpu_expanded);
  hipFree(*gpu_A);
  hipFree(*gpu_B);
}

/* End of code generation (meanImgFiltCZI.cu) */

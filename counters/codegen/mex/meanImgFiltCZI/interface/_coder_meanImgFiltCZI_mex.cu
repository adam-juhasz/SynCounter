/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_meanImgFiltCZI_mex.cu
 *
 * Code generation for function '_coder_meanImgFiltCZI_mex'
 *
 */

/* Include files */
#include "meanImgFiltCZI.h"
#include "_coder_meanImgFiltCZI_mex.h"
#include "meanImgFiltCZI_terminate.h"
#include "_coder_meanImgFiltCZI_api.h"
#include "meanImgFiltCZI_initialize.h"
#include "meanImgFiltCZI_data.h"

/* Function Declarations */
static void meanImgFiltCZI_mexFunction(int32_T nlhs, mxArray *plhs[1], int32_T
  nrhs, const mxArray *prhs[1]);

/* Function Definitions */
static void meanImgFiltCZI_mexFunction(int32_T nlhs, mxArray *plhs[1], int32_T
  nrhs, const mxArray *prhs[1])
{
  const mxArray *outputs[1];

  /* Check for proper number of arguments. */
  if (nrhs != 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 1, 4, 14, "meanImgFiltCZI");
  }

  if (nlhs > 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 14,
                        "meanImgFiltCZI");
  }

  /* Call the function. */
  meanImgFiltCZI_api(prhs, nlhs, outputs);

  /* Copy over outputs to the caller. */
  emlrtReturnArrays(1, plhs, outputs);
}

void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs, const mxArray
                 *prhs[])
{
  mexAtExit(meanImgFiltCZI_atexit);

  /* Module initialization. */
  meanImgFiltCZI_initialize();

  /* Dispatch the entry-point. */
  meanImgFiltCZI_mexFunction(nlhs, plhs, nrhs, prhs);

  /* Module termination. */
  meanImgFiltCZI_terminate();
}

emlrtCTX mexFunctionCreateRootTLS()
{
  emlrtCreateRootTLS(&emlrtRootTLSGlobal, &emlrtContextGlobal, NULL, 1);
  return emlrtRootTLSGlobal;
}

/* End of code generation (_coder_meanImgFiltCZI_mex.cu) */

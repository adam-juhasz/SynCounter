/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_meanImgFiltCZI_api.cu
 *
 * Code generation for function '_coder_meanImgFiltCZI_api'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "meanImgFiltCZI.h"
#include "_coder_meanImgFiltCZI_api.h"
#include "meanImgFiltCZI_data.h"

/* Function Declarations */
static real_T (*b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[1048576];
static real_T (*c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier *
  msgId))[1048576];
static real_T (*emlrt_marshallIn(const mxArray *A, const char_T *identifier))
  [1048576];
static const mxArray *emlrt_marshallOut(const real_T u[1048576]);

/* Function Definitions */
static real_T (*b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[1048576]
{
  real_T (*y)[1048576];
  y = c_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}
  static real_T (*c_emlrt_marshallIn(const mxArray *src, const
  emlrtMsgIdentifier *msgId))[1048576]
{
  real_T (*ret)[1048576];
  static const int32_T dims[2] = { 1024, 1024 };

  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, (const emlrtMsgIdentifier *)msgId,
    src, "double", false, 2U, *(int32_T (*)[2])&dims[0]);
  ret = (real_T (*)[1048576])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T (*emlrt_marshallIn(const mxArray *A, const char_T *identifier))
  [1048576]
{
  real_T (*y)[1048576];
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char *>(identifier);
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(A), &thisId);
  emlrtDestroyArray(&A);
  return y;
}
  static const mxArray *emlrt_marshallOut(const real_T u[1048576])
{
  const mxArray *y;
  const mxArray *m0;
  static const int32_T iv0[2] = { 0, 0 };

  static const int32_T iv1[2] = { 1024, 1024 };

  y = NULL;
  m0 = emlrtCreateNumericArray(2, iv0, mxDOUBLE_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m0, (void *)&u[0]);
  emlrtSetDimensions((mxArray *)m0, *(int32_T (*)[2])&iv1[0], 2);
  emlrtAssign(&y, m0);
  return y;
}

void meanImgFiltCZI_api(const mxArray * const prhs[1], int32_T, const mxArray
  *plhs[1])
{
  real_T (*B)[1048576];
  real_T (*A)[1048576];
  B = (real_T (*)[1048576])mxMalloc(sizeof(real_T [1048576]));

  /* Marshall function inputs */
  A = emlrt_marshallIn(emlrtAlias(prhs[0]), "A");

  /* Invoke the target function */
  meanImgFiltCZI(*A, *B);

  /* Marshall function outputs */
  plhs[0] = emlrt_marshallOut(*B);
}

/* End of code generation (_coder_meanImgFiltCZI_api.cu) */

#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * meanImgFiltLSM.cu
 *
 * Code generation for function 'meanImgFiltLSM'
 *
 */

/* Include files */
#include "MWCudaDimUtility.h"
#include "rt_nonfinite.h"
#include "meanImgFiltLSM.h"

/* Function Declarations */
static __global__ void meanImgFiltLSM_kernel1(real_T expanded[509796]);
static __global__ void meanImgFiltLSM_kernel2(const real_T A[506944], real_T
  expanded[509796]);
static __global__ void meanImgFiltLSM_kernel3(real_T expanded[509796], real_T B
  [506944]);

/* Function Definitions */
static __global__ __launch_bounds__(512, 1) void meanImgFiltLSM_kernel1(real_T
  expanded[509796])
{
  uint32_T threadId;
  int32_T ocol;
  threadId = (uint32_T)mwGetGlobalThreadIndex();
  ocol = (int32_T)threadId;
  if (ocol < 509796) {
    expanded[ocol] = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void meanImgFiltLSM_kernel2(const
  real_T A[506944], real_T expanded[509796])
{
  uint32_T threadId;
  int32_T k;
  int32_T ocol;
  threadId = (uint32_T)mwGetGlobalThreadIndex();
  k = (int32_T)(threadId % 712U);
  ocol = (int32_T)((threadId - (uint32_T)k) / 712U);
  if (ocol < 712) {
    expanded[(k + 714 * (1 + ocol)) + 1] = A[k + 712 * ocol];
  }
}

static __global__ __launch_bounds__(1024, 1) void meanImgFiltLSM_kernel3(real_T
  expanded[509796], real_T B[506944])
{
  real_T y;
  int32_T orow;
  int32_T ocol;
  int32_T k;
  __shared__ real_T expanded_shared[1156];
  int32_T baseR;
  int32_T srow;
  int32_T strideRow;
  int32_T scol;
  int32_T strideCol;
  int32_T y_idx;
  int32_T baseC;
  int32_T x_idx;
  ocol = mwGetGlobalThreadIndexInYDimension();
  orow = mwGetGlobalThreadIndexInXDimension();
  baseR = orow;
  srow = (int32_T)threadIdx.x;
  strideRow = (int32_T)blockDim.x;
  scol = (int32_T)threadIdx.y;
  strideCol = (int32_T)blockDim.y;
  for (y_idx = srow; y_idx <= 33; y_idx += strideRow) {
    baseC = ocol;
    for (x_idx = scol; x_idx <= 33; x_idx += strideCol) {
      if (((int32_T)(((int32_T)(baseR >= 0)) && ((int32_T)(baseR < 714)))) &&
          ((int32_T)(((int32_T)(baseC >= 0)) && ((int32_T)(baseC < 714))))) {
        expanded_shared[y_idx + 34 * x_idx] = expanded[714 * baseC + baseR];
      } else {
        expanded_shared[y_idx + 34 * x_idx] = 0.0;
      }

      baseC += strideCol;
    }

    baseR += strideRow;
  }

  __syncthreads();
  if (((int32_T)(ocol < 712)) && ((int32_T)(orow < 712))) {
    y = expanded_shared[((int32_T)threadIdx.x +
                         (mwGetGlobalThreadIndexInXDimension() -
                          mwGetGlobalThreadIndexInXDimension())) + 34 *
      ((int32_T)threadIdx.y + (mwGetGlobalThreadIndexInYDimension() -
        mwGetGlobalThreadIndexInYDimension()))];
    for (k = 0; k < 8; k++) {
      y += expanded_shared[((int32_T)threadIdx.x + (((k + 1) % 3 +
        mwGetGlobalThreadIndexInXDimension()) -
        mwGetGlobalThreadIndexInXDimension())) + 34 * ((int32_T)threadIdx.y +
        (((k + 1) / 3 + mwGetGlobalThreadIndexInYDimension()) -
         mwGetGlobalThreadIndexInYDimension()))];
    }

    B[orow + 712 * ocol] = y / 9.0;
  }
}

void meanImgFiltLSM(const real_T A[506944], real_T B[506944])
{
  real_T (*gpu_expanded)[509796];
  real_T (*gpu_A)[506944];
  real_T (*gpu_B)[506944];
  hipMalloc(&gpu_B, 4055552ULL);
  hipMalloc(&gpu_A, 4055552ULL);
  hipMalloc(&gpu_expanded, 4078368ULL);
  meanImgFiltLSM_kernel1<<<dim3(996U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_expanded);
  hipMemcpy(gpu_A, (void *)&A[0], 4055552ULL, hipMemcpyHostToDevice);
  meanImgFiltLSM_kernel2<<<dim3(991U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_A,
    *gpu_expanded);
  meanImgFiltLSM_kernel3<<<dim3(23U, 23U, 1U), dim3(32U, 32U, 1U)>>>
    (*gpu_expanded, *gpu_B);
  hipMemcpy(&B[0], gpu_B, 4055552ULL, hipMemcpyDeviceToHost);
  hipFree(*gpu_expanded);
  hipFree(*gpu_A);
  hipFree(*gpu_B);
}

/* End of code generation (meanImgFiltLSM.cu) */

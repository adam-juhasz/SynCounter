/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_meanImgFiltLSM_api.cu
 *
 * Code generation for function '_coder_meanImgFiltLSM_api'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "meanImgFiltLSM.h"
#include "_coder_meanImgFiltLSM_api.h"
#include "meanImgFiltLSM_data.h"

/* Function Declarations */
static real_T (*b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[506944];
static real_T (*c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier *
  msgId))[506944];
static real_T (*emlrt_marshallIn(const mxArray *A, const char_T *identifier))
  [506944];
static const mxArray *emlrt_marshallOut(const real_T u[506944]);

/* Function Definitions */
static real_T (*b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[506944]
{
  real_T (*y)[506944];
  y = c_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}
  static real_T (*c_emlrt_marshallIn(const mxArray *src, const
  emlrtMsgIdentifier *msgId))[506944]
{
  real_T (*ret)[506944];
  static const int32_T dims[2] = { 712, 712 };

  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, (const emlrtMsgIdentifier *)msgId,
    src, "double", false, 2U, *(int32_T (*)[2])&dims[0]);
  ret = (real_T (*)[506944])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T (*emlrt_marshallIn(const mxArray *A, const char_T *identifier))
  [506944]
{
  real_T (*y)[506944];
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char *>(identifier);
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(A), &thisId);
  emlrtDestroyArray(&A);
  return y;
}
  static const mxArray *emlrt_marshallOut(const real_T u[506944])
{
  const mxArray *y;
  const mxArray *m0;
  static const int32_T iv0[2] = { 0, 0 };

  static const int32_T iv1[2] = { 712, 712 };

  y = NULL;
  m0 = emlrtCreateNumericArray(2, iv0, mxDOUBLE_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m0, (void *)&u[0]);
  emlrtSetDimensions((mxArray *)m0, *(int32_T (*)[2])&iv1[0], 2);
  emlrtAssign(&y, m0);
  return y;
}

void meanImgFiltLSM_api(const mxArray * const prhs[1], int32_T, const mxArray
  *plhs[1])
{
  real_T (*B)[506944];
  real_T (*A)[506944];
  B = (real_T (*)[506944])mxMalloc(sizeof(real_T [506944]));

  /* Marshall function inputs */
  A = emlrt_marshallIn(emlrtAlias(prhs[0]), "A");

  /* Invoke the target function */
  meanImgFiltLSM(*A, *B);

  /* Marshall function outputs */
  plhs[0] = emlrt_marshallOut(*B);
}

/* End of code generation (_coder_meanImgFiltLSM_api.cu) */

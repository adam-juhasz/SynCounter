/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_meanImgFiltLSM_mex.cu
 *
 * Code generation for function '_coder_meanImgFiltLSM_mex'
 *
 */

/* Include files */
#include "meanImgFiltLSM.h"
#include "_coder_meanImgFiltLSM_mex.h"
#include "meanImgFiltLSM_terminate.h"
#include "_coder_meanImgFiltLSM_api.h"
#include "meanImgFiltLSM_initialize.h"
#include "meanImgFiltLSM_data.h"

/* Function Declarations */
static void meanImgFiltLSM_mexFunction(int32_T nlhs, mxArray *plhs[1], int32_T
  nrhs, const mxArray *prhs[1]);

/* Function Definitions */
static void meanImgFiltLSM_mexFunction(int32_T nlhs, mxArray *plhs[1], int32_T
  nrhs, const mxArray *prhs[1])
{
  const mxArray *outputs[1];

  /* Check for proper number of arguments. */
  if (nrhs != 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 1, 4, 14, "meanImgFiltLSM");
  }

  if (nlhs > 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 14,
                        "meanImgFiltLSM");
  }

  /* Call the function. */
  meanImgFiltLSM_api(prhs, nlhs, outputs);

  /* Copy over outputs to the caller. */
  emlrtReturnArrays(1, plhs, outputs);
}

void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs, const mxArray
                 *prhs[])
{
  mexAtExit(meanImgFiltLSM_atexit);

  /* Module initialization. */
  meanImgFiltLSM_initialize();

  /* Dispatch the entry-point. */
  meanImgFiltLSM_mexFunction(nlhs, plhs, nrhs, prhs);

  /* Module termination. */
  meanImgFiltLSM_terminate();
}

emlrtCTX mexFunctionCreateRootTLS()
{
  emlrtCreateRootTLS(&emlrtRootTLSGlobal, &emlrtContextGlobal, NULL, 1);
  return emlrtRootTLSGlobal;
}

/* End of code generation (_coder_meanImgFiltLSM_mex.cu) */
